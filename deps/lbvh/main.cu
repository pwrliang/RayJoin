#include "hip/hip_runtime.h"
#include "lbvh.cuh"
#include <random>
#include <vector>
#include <thrust/random.h>

struct aabb_getter
{
    __device__
    lbvh::aabb<float> operator()(const float4 f) const noexcept
    {
        lbvh::aabb<float> retval;
        retval.upper = f;
        retval.lower = f;
        return retval;
    }
};
struct distance_calculator
{
    __device__
    float operator()(const float4 point, const float4 object) const noexcept
    {
        return (point.x - object.x) * (point.x - object.x) +
               (point.y - object.y) * (point.y - object.y) +
               (point.z - object.z) * (point.z - object.z);
    }
};

int main()
{
    constexpr std::size_t N=10;
    std::vector<float4> ps(N);

    std::mt19937 mt(123456789);
    std::uniform_real_distribution<float> uni(0.0, 1.0);

    for(auto& p : ps)
    {
        p.x = uni(mt);
        p.y = uni(mt);
        p.z = uni(mt);
    }

    lbvh::bvh<float, float4, aabb_getter> bvh(ps.begin(), ps.end(), true);

    const auto bvh_dev = bvh.get_device_repr();

    std::cout << "testing query_device:overlap ...\n";
    thrust::for_each(thrust::device,
        thrust::make_counting_iterator<std::size_t>(0),
        thrust::make_counting_iterator<std::size_t>(N),
        [bvh_dev] __device__ (std::size_t idx) {
            unsigned int buffer[10];
            const auto self = bvh_dev.objects[idx];
            const float  dr = 0.1f;
            for(std::size_t i=1; i<10; ++i)
            {
                for(unsigned int j=0; j<10; ++j)
                {
                    buffer[j] = 0xFFFFFFFF;
                }
                const float r = dr * i;
                lbvh::aabb<float> query_box;
                query_box.lower = make_float4(self.x-r, self.y-r, self.z-r, 0);
                query_box.upper = make_float4(self.x+r, self.y+r, self.z+r, 0);
                const auto num_found = lbvh::query_device(
                        bvh_dev, lbvh::overlaps(query_box), buffer, 10);

                for(unsigned int j=0; j<10; ++j)
                {
                    const auto jdx    = buffer[j];
                    if(j >= num_found)
                    {
                        assert(jdx == 0xFFFFFFFF);
                        continue;
                    }
                    else
                    {
                        assert(jdx != 0xFFFFFFFF);
                        assert(jdx < bvh_dev.num_objects);
                    }
                    const auto other  = bvh_dev.objects[jdx];
                    assert(fabsf(self.x - other.x) < r); // check coordinates
                    assert(fabsf(self.y - other.y) < r); // are in the range
                    assert(fabsf(self.z - other.z) < r); // of query box
                }
            }
            return ;
        });

    std::cout << "testing query_device:nearest_neighbor ...\n";
    thrust::for_each(thrust::device,
        thrust::make_counting_iterator<unsigned int>(0),
        thrust::make_counting_iterator<unsigned int>(N),
        [bvh_dev] __device__ (const unsigned int idx) {
            const auto self = bvh_dev.objects[idx];
            const auto nest = lbvh::query_device(bvh_dev, lbvh::nearest(self),
                                                 distance_calculator());
            assert(nest.first != 0xFFFFFFFF);
            const auto other   = bvh_dev.objects[nest.first];
            // of course, the nearest object is itself.
            assert(nest.second == 0.0f);
            assert(self.x == other.x);
            assert(self.y == other.y);
            assert(self.z == other.z);
            return ;
       });

    thrust::device_vector<float4> random_points(N);
    thrust::transform(
        thrust::make_counting_iterator<unsigned int>(0),
        thrust::make_counting_iterator<unsigned int>(N),
        random_points.begin(), [] __device__(const unsigned int idx) {
            thrust::default_random_engine rand;
            thrust::uniform_real_distribution<float> uni(0.0f, 1.0f);
            rand.discard(idx);
            const float x = uni(rand);
            const float y = uni(rand);
            const float z = uni(rand);
            return make_float4(x, y, z, 0);
        });

    thrust::for_each(random_points.begin(), random_points.end(),
        [bvh_dev] __device__ (const float4 pos) {
            const auto calc = distance_calculator();
            const auto nest = lbvh::query_device(bvh_dev, lbvh::nearest(pos), calc);
            assert(nest.first != 0xFFFFFFFF);

            for(unsigned int i=0; i<bvh_dev.num_objects; ++i)
            {
                const auto dist = calc(bvh_dev.objects[i], pos);
                if(i == nest.first)
                {
                    assert(dist == nest.second);
                }
                else
                {
                    assert(dist >= nest.second);
                }
            }
            return ;
        });

    std::cout << "testing query_host:overlap ...\n";
    {
        for(std::size_t i=0; i<10; ++i)
        {
            const auto self = bvh.objects_host()[i];
            const float dr = 0.1f;
            for(unsigned int cnt=1; cnt<10; ++cnt)
            {
                const float r = dr * cnt;
                lbvh::aabb<float> query_box;
                query_box.lower = make_float4(self.x-r, self.y-r, self.z-r, 0);
                query_box.upper = make_float4(self.x+r, self.y+r, self.z+r, 0);

                std::vector<std::size_t> buffer;
                const auto num_found = lbvh::query_host(bvh,
                        lbvh::overlaps(query_box), std::back_inserter(buffer));

                for(unsigned int jdx : buffer)
                {
                    assert(jdx < bvh.objects_host().size());

                    const auto other  = bvh.objects_host()[jdx];
                    assert(fabsf(self.x - other.x) < r); // check coordinates
                    assert(fabsf(self.y - other.y) < r); // are in the range
                    assert(fabsf(self.z - other.z) < r); // of query box
                }
            }
        }
    }
    return 0;
}
