#include "hip/hip_runtime.h"
#include <memory>
#include <random>

#include "app/lsi_grid.h"
#include "app/lsi_lbvh.h"
#include "app/lsi_rt.h"
#include "app/pip_grid.h"
#include "app/pip_lbvh.h"
#include "app/pip_rt.h"
#include "map/planar_graph.h"
#include "rt/primitive.h"
#include "run_query.cuh"
#include "tree/primtive.h"
#include "util/array_view.h"
#include "util/helpers.h"
#include "util/timer.h"

namespace rayjoin {

template <typename CONTEXT_T>
void CheckPIPResult(
    CONTEXT_T& ctx, QueryConfig config,
    const thrust::device_vector<typename CONTEXT_T::map_t::point_t>& points,
    const thrust::device_vector<index_t>& eids) {
  auto& stream = ctx.get_stream();
  auto grid = std::make_shared<UniformGrid>(config.grid_size);
  int base_map_id = 0;
  grid->AddMapToGrid(ctx, base_map_id, !config.profiling.empty());
  PIPGrid<CONTEXT_T> pip_grid(ctx, grid);

  LOG(INFO) << "Checking point in polygon";
  pip_grid.Query(stream, base_map_id, points);
  stream.Sync();

  pinned_vector<typename CONTEXT_T::map_t::point_t> h_points = points;
  pinned_vector<index_t> closest_eids_ans = pip_grid.get_closest_eids();
  pinned_vector<index_t> closest_eids_res = eids;
  size_t n_diff = 0;
  size_t n_points = h_points.size();

  CHECK_EQ(closest_eids_res.size(), closest_eids_ans.size());

  auto base_map = ctx.get_map(base_map_id);
  auto scaling = ctx.get_scaling();

  for (size_t point_idx = 0; point_idx < n_points; point_idx++) {
    auto closest_eid_ans = closest_eids_ans[point_idx];
    auto closest_eid_res = closest_eids_res[point_idx];
    // different eid does not mean wrong answer because there are two edges
    // having same coordinates but different eid
    if (closest_eid_res != closest_eid_ans) {
      auto not_hit = std::numeric_limits<index_t>::max();
      auto p = ctx.get_planar_graph(base_map_id)->points[point_idx];
      auto scaled_p = h_points[point_idx];
      bool diff = false;
      std::string ep_ans = "miss";
      std::string ep_res = "miss";
      std::string scaled_ep_ans = "miss";
      std::string scaled_ep_res = "miss";

      if (closest_eid_ans != not_hit) {
        ep_ans = base_map->EndpointsToString(closest_eid_ans, scaling);
        scaled_ep_ans = base_map->ScaledEndpointsToString(closest_eid_ans);
      }

      if (closest_eid_res != not_hit) {
        ep_res = base_map->EndpointsToString(closest_eid_res, scaling);
        scaled_ep_res = base_map->ScaledEndpointsToString(closest_eid_res);
      }

      if (scaled_ep_res != scaled_ep_ans) {
        diff = true;
      }

      if (diff && n_diff < 10) {
        printf("point %lu (%.8lf, %.8lf) ans %u, res %u %s %s\n", point_idx,
               p.x, p.y, closest_eid_ans, closest_eid_res, ep_ans.c_str(),
               ep_res.c_str());
        printf("scaled point %lu (%ld, %ld) ans %u, res %u %s %s\n", point_idx,
               scaled_p.x, scaled_p.y, closest_eid_ans, closest_eid_res,
               scaled_ep_ans.c_str(), scaled_ep_res.c_str());
      }

      if (diff) {
        n_diff++;
      }
    }
  }
  if (n_diff != 0) {
    LOG(ERROR) << "Map: " << base_map_id << " Total points: " << n_points
               << " n diff: " << n_diff
               << " Error rate: " << (double) n_diff / n_points * 100 << " %";
  } else {
    LOG(INFO) << "Map: " << base_map_id << " passed check";
  }
}

template <typename CONTEXT_T>
std::shared_ptr<typename CONTEXT_T::map_t> GenerateLSIQueries(
    const QueryConfig& config, CONTEXT_T& ctx) {
  using coord_t = typename CONTEXT_T::coord_t;
  using coefficient_t = typename CONTEXT_T::coefficient_t;
  using point_t = typename cuda_vec<coord_t>::type_2d;
  using edge_t = dev::Edge<coefficient_t>;
  auto bb = ctx.get_bounding_box();
  auto& stream = ctx.get_stream();
  auto scaling = ctx.get_scaling();
  size_t ne = config.gen_n;
  auto max_t = config.gen_t;
  auto seed = config.random_seed;
  std::random_device rd;  // obtain a random number from hardware
  std::mt19937 gen(seed == 0 ? rd() : seed);  // seed the generator
  std::uniform_real_distribution<> dist_x(bb.min_x, bb.max_x);  // unscaled x,y
  std::uniform_real_distribution<> dist_y(bb.min_y, bb.max_y);
  std::uniform_real_distribution<> dist_t(0, max_t);

  auto query_map = std::make_shared<typename CONTEXT_T::map_t>(1);

  pinned_vector<point_t> points(ne * 2);
  pinned_vector<edge_t> edges(ne);

  for (size_t i = 0; i < ne; i++) {
    auto x1 = dist_x(gen), y1 = dist_y(gen);
    auto x2 = dist_x(gen), y2 = dist_y(gen);
    auto len = sqrt((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1));
    auto d_x = (x2 - x1) / len, d_y = (y2 - y1) / len;
    auto t = dist_t(gen);

    point_t p1{x1, y1};
    point_t p2{x1 + t * d_x, y1 + t * d_y};

    points[2 * i] = p1;
    points[2 * i + 1] = p2;
    edges[i].eid = i;
    edges[i].p1_idx = 2 * i;
    edges[i].p2_idx = 2 * i + 1;
    edges[i].left_polygon_id = edges[i].right_polygon_id = 0;
  }
  query_map->LoadFrom(stream, scaling, points, edges);
  return query_map;
}

template <typename CONTEXT_T>
thrust::host_vector<typename CONTEXT_T::map_t::point_t> GeneratePIPQueries(
    const QueryConfig& config, CONTEXT_T& ctx) {
  using coefficient_t = typename CONTEXT_T::coefficient_t;
  auto bb = ctx.get_bounding_box();
  auto scaling = ctx.get_scaling();
  auto seed = config.random_seed;
  std::random_device rd;  // obtain a random number from hardware
  std::mt19937 gen(seed == 0 ? rd() : seed);  // seed the generator
  std::uniform_real_distribution<> dist_x(bb.min_x, bb.max_x);  // unscaled x,y
  std::uniform_real_distribution<> dist_y(bb.min_y, bb.max_y);

  thrust::host_vector<typename CONTEXT_T::map_t::point_t> points;

  for (size_t i = 0; i < config.gen_n; i++) {
    auto x = dist_x(gen), y = dist_y(gen);

    points.push_back({scaling.ScaleX(x), scaling.ScaleY(y)});
  }

  return points;
}

void RunLSIQuery(const QueryConfig& config) {
  using context_t = Context<coord_t, coefficient_t>;
  using internal_coord_t = typename context_t::internal_coord_t;

  timer_start();
  timer_next("Read map");
  auto base_map = load_from<coord_t>(config.map_path, config.serialize_prefix);
  int base_map_id = 0, query_map_id = 1;

  if (config.sample == "edges") {
    LOG(INFO) << "Sampling edges from map, sample rate: " << config.sample_rate
              << ", seed: " << config.random_seed;
    base_map =
        sample_edges_from(base_map, config.sample_rate, config.random_seed);
  } else if (config.sample == "map") {
    LOG(INFO) << "Sampling map, sample rate: " << config.sample_rate
              << ", seed: " << config.random_seed;
    base_map =
        sample_map_from(base_map, config.sample_rate, config.random_seed);
  }

  timer_next("Create Context");
  context_t ctx(base_map);
  Stream& stream = ctx.get_stream();
  LSI<context_t>* lsi;

  timer_next("Generate Workloads");
  auto query_map = GenerateLSIQueries(config, ctx);
  ctx.set_map(query_map_id, query_map);

  timer_next("Create App");
  if (config.mode == "grid") {
    auto grid = std::make_shared<UniformGrid>(config.grid_size);

    lsi = new LSIGrid<context_t>(ctx, grid);
    dynamic_cast<LSIGrid<context_t>*>(lsi)->set_load_balancing(config.lb);
  } else if (config.mode == "lbvh") {
    lsi = new LSILBVH<context_t>(ctx);
  } else if (config.mode == "rt") {
    auto rt_engine = std::make_shared<RTEngine>();
    RTConfig rt_config = get_default_rt_config(config.exec_root);

    rt_engine->Init(rt_config);
    lsi = new LSIRT<context_t>(ctx, rt_engine);
  } else {
    LOG(FATAL) << "Invalid index type: " << config.mode;
  }

  timer_next("Init");
  auto d_base_map = ctx.get_map(base_map_id)->DeviceObject();
  auto d_query_map = ctx.get_map(query_map_id)->DeviceObject();
  const auto& scaling = ctx.get_scaling();
  size_t queue_cap =
      (d_base_map.get_edges_num() + d_query_map.get_edges_num()) *
      config.xsect_factor;
  LOG(INFO) << "Queue capacity: " << queue_cap;
  lsi->Init(queue_cap);

  timer_next("Build Index");
  if (config.mode == "grid") {
    auto lsi_grid = dynamic_cast<LSIGrid<context_t>*>(lsi)->get_grid();

    lsi_grid->AddMapsToGrid(ctx, !config.profiling.empty());
  } else if (config.mode == "rt") {
    auto lsi_rt = dynamic_cast<LSIRT<context_t>*>(lsi);
    thrust::device_vector<OptixAabb> aabbs;
    auto eid_range =
        std::make_shared<thrust::device_vector<thrust::pair<size_t, size_t>>>();
    QueryConfigRT query_config;
    auto rt_engine = lsi_rt->get_rt_engine();
    auto win_size = config.win;
    auto area_enlarge = config.enlarge;

    FillPrimitivesGroup(stream, d_base_map, scaling, win_size, area_enlarge,
                        aabbs, *eid_range);

    query_config.fau = config.fau;
    query_config.rounding_iter = config.rounding_iter;
    query_config.handle =
        rt_engine->BuildAccelCustom(stream, ArrayView<OptixAabb>(aabbs));
    query_config.eid_range = eid_range;

    lsi_rt->set_config(query_config);
  } else if (config.mode == "lbvh") {
    auto lsi_lbvh = dynamic_cast<LSILBVH<context_t>*>(lsi);
    QueryConfigLBVH query_config;
    pinned_vector<segment> primitives;
    auto bvh = std::make_shared<lbvh::bvh<float, segment, aabb_getter>>();

    FillPrimitivesLBVH(stream, d_base_map, scaling, primitives);
    stream.Sync();
    bvh->assign(primitives);
    bvh->construct(!config.profiling.empty());

    query_config.lbvh = bvh;
    query_config.profiling = !config.profiling.empty();
    lsi_lbvh->set_config(query_config);
  }

  timer_next("Warmup");
  for (int i = 0; i < config.warmup; i++) {
    lsi->Query(query_map_id);
  }

  timer_next("Query", config.repeat);
  ArrayView<typename LSI<context_t>::xsect_t> d_xsects;

  for (int i = 0; i < config.repeat; i++) {
    LOG(INFO) << "Iter: " << i;
    d_xsects = lsi->Query(query_map_id);
  }

  LOG(INFO) << "Query: " << config.gen_n
            << " Intersections: " << d_xsects.size()
            << " Selective: " << (double) d_xsects.size() / config.gen_n
            << " Queue Load Factor: " << (double) d_xsects.size() / queue_cap;

  timer_next("Cleanup");

  delete lsi;
  timer_end();
}

void RunPIPQuery(const QueryConfig& config) {
  using context_t = Context<coord_t, coefficient_t>;
  using internal_coord_t = typename context_t::internal_coord_t;
  using point_t = typename context_t::map_t::point_t;

  timer_start();
  timer_next("Read map");
  auto base_map = load_from<coord_t>(config.map_path, config.serialize_prefix);
  int base_map_id = 0, query_map_id = 1;

  if (config.sample == "edges") {
    LOG(INFO) << "Sampling edges from map, sample rate: " << config.sample_rate
              << ", seed: " << config.random_seed;
    base_map =
        sample_edges_from(base_map, config.sample_rate, config.random_seed);
  } else if (config.sample == "map") {
    LOG(INFO) << "Sampling map, sample rate: " << config.sample_rate
              << ", seed: " << config.random_seed;
    base_map =
        sample_map_from(base_map, config.sample_rate, config.random_seed);
  }

  timer_next("Create Context");
  context_t ctx(base_map);
  PIP<context_t>* pip;
  Stream& stream = ctx.get_stream();

  timer_next("Generate Workloads");
  thrust::device_vector<point_t> query_points =
      rayjoin::GeneratePIPQueries(config, ctx);
  ArrayView<typename context_t::map_t::point_t> d_query_points(query_points);

  timer_next("Create App");
  if (config.mode == "grid") {
    auto grid = std::make_shared<UniformGrid>(config.grid_size);

    pip = new PIPGrid<context_t>(ctx, grid);
  } else if (config.mode == "rt") {
    auto rt_engine = std::make_shared<RTEngine>();
    RTConfig rt_config = get_default_rt_config(config.exec_root);

    rt_engine->Init(rt_config);
    pip = new PIPRT<context_t>(ctx, rt_engine);
  } else if (config.mode == "lbvh") {
    pip = new PIPLBVH<context_t>(ctx);
  } else {
    LOG(FATAL) << "Invalid index type: " << config.mode;
  }

  timer_next("Init");
  auto d_base_map = ctx.get_map(base_map_id)->DeviceObject();
  auto d_query_map = ctx.get_map(query_map_id)->DeviceObject();
  const auto& scaling = ctx.get_scaling();
  pip->Init(query_points.size());

  timer_next("Build Index");
  if (config.mode == "grid") {
    auto grid = dynamic_cast<PIPGrid<context_t>*>(pip)->get_grid();

    grid->AddMapToGrid(ctx, 0, !config.profiling.empty());
  } else if (config.mode == "rt") {
    auto pip_rt = dynamic_cast<PIPRT<context_t>*>(pip);
    auto rt_engine = pip_rt->get_rt_engine();
    thrust::device_vector<OptixAabb> aabbs;
    auto eid_range =
        std::make_shared<thrust::device_vector<thrust::pair<size_t, size_t>>>();
    auto ne = d_base_map.get_edges_num();

    FillPrimitivesGroup(stream, d_base_map, scaling, config.win, config.enlarge,
                        aabbs, *eid_range);

    ArrayView<OptixAabb> d_aabbs(aabbs);

    QueryConfigRT pip_config;

    pip_config.fau = config.fau;
    pip_config.eid_range = eid_range;
    pip_config.handle = rt_engine->BuildAccelCustom(stream, d_aabbs);

    pip_rt->set_query_config(pip_config);
  } else if (config.mode == "lbvh") {
    auto pip_lbvh = dynamic_cast<PIPLBVH<context_t>*>(pip);
    QueryConfigLBVH query_config;
    pinned_vector<segment> primitives;
    auto bvh = std::make_shared<lbvh::bvh<float, segment, aabb_getter>>();

    FillPrimitivesLBVH(stream, d_base_map, scaling, primitives);
    stream.Sync();
    bvh->assign(primitives);
    bvh->construct(!config.profiling.empty());

    query_config.lbvh = bvh;
    query_config.profiling = !config.profiling.empty();
    pip_lbvh->set_config(query_config);
  }
  stream.Sync();

  timer_next("Warmup");

  for (int i = 0; i < config.warmup; i++) {
    pip->Query(stream, 0, query_points);
  }
  stream.Sync();

  timer_next("Query", config.repeat);
  for (int i = 0; i < config.repeat; i++) {
    pip->Query(stream, 0, query_points);

    if (i == config.repeat - 1) {
      stream.Sync();
      if (config.check && config.mode != "grid") {
        timer_next("Check");
        CheckPIPResult(ctx, config, query_points, pip->get_closest_eids());
      }
    }
  }

  timer_next("Cleanup");

  delete pip;
  timer_end();
}
}  // namespace rayjoin